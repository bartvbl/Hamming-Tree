#include "hip/hip_runtime.h"
#include "quasiSpinImageGenerator.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <shapeSearch/gpu/types/DeviceMesh.h>
#include <shapeSearch/gpu/types/GPURasterisationSettings.h>
#include <shapeSearch/gpu/types/CudaLaunchDimensions.h>
#include <shapeSearch/gpu/setValue.cuh>
#include <shapeSearch/libraryBuildSettings.h>
#include <shapeSearch/common/types/QSIPrecalculatedSettings.h>

#include "nvidia/shfl_scan.cuh"
#include "nvidia/hip/hip_vector_types.h"
#include "nvidia/hip/hip_runtime_api.h"

#include <assert.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <chrono>
#include <sstream>

#if QSI_PIXEL_DATATYPE == DATATYPE_UNSIGNED_SHORT
const int SHORT_SINGLE_BOTH_MASK = 0x00010001;
const int SHORT_SINGLE_ONE_MASK = 0x00000001;
const int SHORT_SINGLE_FIRST_MASK = 0x00010000;

const int SHORT_DOUBLE_BOTH_MASK = 0x00020002;
const int SHORT_DOUBLE_ONE_MASK = 0x00000002;
const int SHORT_DOUBLE_FIRST_MASK = 0x00020000;
#endif

const int RASTERISATION_WARP_SIZE = 1024;

__device__ __inline__ QSIPrecalculatedSettings calculateRotationSettings(float3 spinImageNormal);

__device__ __inline__ float transformNormalX(QSIPrecalculatedSettings pre_settings, float3 spinImageNormal)
{
	return pre_settings.alignmentProjection_n_ax * spinImageNormal.x + pre_settings.alignmentProjection_n_ay * spinImageNormal.y;
}

__device__ __inline__ float3 transformCoordinate(float3 vertex, GPURasterisationSettings settings)
{
	QSIPrecalculatedSettings spinImageSettings = calculateRotationSettings(settings.spinImageNormal);

	float3 transformedCoordinate = vertex - settings.spinImageVertex;

	float initialTransformedX = transformedCoordinate.x;
	transformedCoordinate.x = spinImageSettings.alignmentProjection_n_ax * transformedCoordinate.x + spinImageSettings.alignmentProjection_n_ay * transformedCoordinate.y;
	transformedCoordinate.y = -spinImageSettings.alignmentProjection_n_ay * initialTransformedX + spinImageSettings.alignmentProjection_n_ax * transformedCoordinate.y;

	// Order matters here
	initialTransformedX = transformedCoordinate.x;
	transformedCoordinate.x = spinImageSettings.alignmentProjection_n_bz * transformedCoordinate.x - spinImageSettings.alignmentProjection_n_bx * transformedCoordinate.z;
	transformedCoordinate.z = spinImageSettings.alignmentProjection_n_bx * initialTransformedX + spinImageSettings.alignmentProjection_n_bz * transformedCoordinate.z;

	return transformedCoordinate;
}

__device__ __inline__ float2 alignWithPositiveX(float2 midLineDirection, float2 vertex)
{
	float2 transformed;
	transformed.x = midLineDirection.x * vertex.x + midLineDirection.y * vertex.y;
	transformed.y = -midLineDirection.y * vertex.x + midLineDirection.x * vertex.y;
	return transformed;
}

__device__ __inline__ float calculateTransformedZCoordinate(GPURasterisationSettings settings, float3 vertex)
{
	QSIPrecalculatedSettings pre_settings = calculateRotationSettings(settings.spinImageNormal);

	// Translate to origin
	vertex -= settings.spinImageVertex;

	// Since we're looking at 2 axis of a 3D normal vector, both axis may be 0.
	// In this first case: if x = 0 and y = 0, z = 1 or z = -1. This is already part of the vertical plane we'd like the rotated
	// coordinates to be in, so nothing needs to be done.
	float transformedX = pre_settings.alignmentProjection_n_ax * vertex.x + pre_settings.alignmentProjection_n_ay * vertex.y;

	// In this second case: if x = 0 and z = 0, y = 1 or y = -1. In both cases the previous step already rotated the vector to the correct
	// direction, and nothing else needs to be done.
	//float transformedZ = settings.alignmentProjection_n_bx * transformedX + settings.alignmentProjection_n_bz * triangleVertices[i].z;
	float transformedZ = pre_settings.alignmentProjection_n_bx * transformedX + pre_settings.alignmentProjection_n_bz * vertex.z;

	// Account for that samples of pixels are considered centred. We thus need to add a distance value to each z-coordinate.
	//transformedZ -= settings.nudgeDistance;

	return transformedZ;
}

__device__ __inline__ void rasteriseRow(int pixelBaseIndex, newSpinImagePixelType* descriptorArray, unsigned int pixelStart, unsigned int pixelEnd, const unsigned int singleMask, const unsigned int doubleMask, const unsigned int initialMask)
{
	// First we calculate a base pointer for the first short value that should be updated
	newSpinImagePixelType* rowStartPointer = descriptorArray + pixelBaseIndex;
	// Next, since atomicAdd() requires an integer pointer, we force a cast to an integer pointer
	// while preserving the address of the original
	unsigned int* jobBasePixelPointer = (unsigned int*)((void*)(rowStartPointer));
	// We need an aligned pointer for atomicAdd, so we zero the final two bits of the pointer.
	// We use shifts because the actual size of uintprt_t is not known. Could be 40-bit.
	unsigned int* jobAlignedPointer = (unsigned int*)((((uintptr_t)jobBasePixelPointer) >> 2) << 2);

	int pixelCount = pixelEnd - pixelStart;

	// Zero pixel counts and unchecked pixel ranges can still exist at this point.
	// The initial loop was meant to filter them out
	// pixelEnd is not clamped, resulting in a negative overflow
	// The equals check ensures zero width ranges are filtered.
	if(pixelEnd <= pixelStart)
	{
		return;
	}

	assert((unsigned long) (jobBasePixelPointer) - (unsigned long) (jobAlignedPointer) == 0 || (unsigned long) (jobBasePixelPointer) - (unsigned long) (jobAlignedPointer) == 2);

	unsigned int currentMask = doubleMask;

	// In 1 / 2 cases, the aligned pointer will have been moved back.
	// We thus need to update the latter short in the slot, and move on to the next.
	if(jobAlignedPointer < jobBasePixelPointer)
	{
		currentMask = initialMask;
		// Needed to keep the loop going on case another pixel update needs to be done
		pixelCount++;
	// The other special scenario is a single pixel at the start. This only occurs when the base pointer equals the
	// aligned pointer and the total pixel count is 1.
	} else if(pixelCount == 1) {
		currentMask = singleMask;
	}

	unsigned int jobPointerOffset = 0;

	// We need the rounding down behaviour of division here to calculate number of "full" updates
	while (pixelCount > 0)
	{
		unsigned int* updateAddress = jobAlignedPointer + jobPointerOffset;

		atomicAdd(updateAddress, currentMask);

		pixelCount -= 2;
		jobPointerOffset++;

		currentMask = pixelCount == 1 ? singleMask : doubleMask;
	}
}

__forceinline__ __device__ unsigned lane_id()
{
	unsigned ret; 
	asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __inline__ QSIPrecalculatedSettings calculateRotationSettings(float3 spinImageNormal) {

// Calculating the transformation factors
	QSIPrecalculatedSettings pre_settings;

	float2 sineCosineAlpha = normalize(make_float2(spinImageNormal.x, spinImageNormal.y));

	bool is_n_a_not_zero = !((abs(spinImageNormal.x) < MAX_EQUIVALENCE_ROUNDING_ERROR) && (abs(spinImageNormal.y) < MAX_EQUIVALENCE_ROUNDING_ERROR));

	if (is_n_a_not_zero)
	{
		pre_settings.alignmentProjection_n_ax = sineCosineAlpha.x;
		pre_settings.alignmentProjection_n_ay = sineCosineAlpha.y;
	}
	else
	{
		// Leave values unchanged
		pre_settings.alignmentProjection_n_ax = 1;
		pre_settings.alignmentProjection_n_ay = 0;
	}

	float transformedNormalX = transformNormalX(pre_settings, spinImageNormal);

	float2 sineCosineBeta = normalize(make_float2(transformedNormalX, spinImageNormal.z));

	bool is_n_b_not_zero = !((abs(transformedNormalX) < MAX_EQUIVALENCE_ROUNDING_ERROR) && (abs(spinImageNormal.z) < MAX_EQUIVALENCE_ROUNDING_ERROR));

	if (is_n_b_not_zero)
	{
		pre_settings.alignmentProjection_n_bx = sineCosineBeta.x;
		pre_settings.alignmentProjection_n_bz = sineCosineBeta.y; // discrepancy between axis here is because we are using a 2D vector on 3D axis.
	}
	else
	{
		// Leave input values unchanged
		pre_settings.alignmentProjection_n_bx = 1;
		pre_settings.alignmentProjection_n_bz = 0;
	}

	return pre_settings;
}

__device__ __inline__ void rasteriseTriangle(
#if ENABLE_SHARED_MEMORY_IMAGE
		newSpinImagePixelType* sharedDescriptorArray,
#else
		array<newSpinImagePixelType> descriptors,
#endif
		float3 vertices[3], GPURasterisationSettings settings)
{
	vertices[0] = transformCoordinate(vertices[0], settings);
	vertices[1] = transformCoordinate(vertices[1], settings);
	vertices[2] = transformCoordinate(vertices[2], settings);

	float3 minVector = { 0, 0, 0 };
	float3 midVector = { 0, 0, 0 };
	float3 maxVector = { 0, 0, 0 };

	float3 deltaMinMid = { 0, 0, 0 };
	float3 deltaMidMax = { 0, 0, 0 };
	float3 deltaMinMax = { 0, 0, 0 };

	// Sort vertices by z-coordinate

	int minIndex = 0;
	int midIndex = 1;
	int maxIndex = 2;
	int _temp;

	if (vertices[minIndex].z > vertices[midIndex].z)
	{
		_temp = minIndex;
		minIndex = midIndex;
		midIndex = _temp;
	}
	if (vertices[minIndex].z > vertices[maxIndex].z)
	{
		_temp = minIndex;
		minIndex = maxIndex;
		maxIndex = _temp;
	}
	if (vertices[midIndex].z > vertices[maxIndex].z)
	{
		_temp = midIndex;
		midIndex = maxIndex;
		maxIndex = _temp;
	}

	minVector = vertices[minIndex];
	midVector = vertices[midIndex];
	maxVector = vertices[maxIndex];

	// Calculate deltas

	deltaMinMid = midVector - minVector;
	deltaMidMax = maxVector - midVector;
	deltaMinMax = maxVector - minVector;

	// Horizontal triangles are most likely not to register, and cause zero divisions, so it's easier to just get rid of them.
	if (deltaMinMax.z < MAX_EQUIVALENCE_ROUNDING_ERROR)
	{
		return;
	}

	float2 minXY = { 0, 0 };
	float2 midXY = { 0, 0 };
	float2 maxXY = { 0, 0 };

	float2 deltaMinMidXY = { 0, 0 };
	float2 deltaMidMaxXY = { 0, 0 };
	float2 deltaMinMaxXY = { 0, 0 };

	int minPixels = 0;
	int maxPixels = 0;

	// Step 6: Calculate centre line
	float centreLineFactor = deltaMinMid.z / deltaMinMax.z;
	float2 centreLineDelta = centreLineFactor * make_float2(deltaMinMax.x, deltaMinMax.y);
	float2 centreLineDirection = centreLineDelta - make_float2(deltaMinMid.x, deltaMinMid.y);
	float2 centreDirection = normalize(centreLineDirection);

	// Step 7: Rotate coordinates around origin
	// From here on out, variable names follow these conventions:
	// - X: physical relative distance to closest point on intersection line
	// - Y: Distance from origin
	minXY = alignWithPositiveX(centreDirection, make_float2(minVector.x, minVector.y));
	midXY = alignWithPositiveX(centreDirection, make_float2(midVector.x, midVector.y));
	maxXY = alignWithPositiveX(centreDirection, make_float2(maxVector.x, maxVector.y));

	deltaMinMidXY = midXY - minXY;
	deltaMidMaxXY = maxXY - midXY;
	deltaMinMaxXY = maxXY - minXY;

	// Step 8: For each row, do interpolation
	minPixels = int(floor(minVector.z /** settings.oneOverPixelSize*/));
	maxPixels = int(floor(maxVector.z /** settings.oneOverPixelSize*/));

	// Ensure we only rasterise within bounds
	minPixels = clamp(minPixels, (-spinImageWidthPixels / 2), (spinImageWidthPixels / 2) - 1);
	maxPixels = clamp(maxPixels, (-spinImageWidthPixels / 2), (spinImageWidthPixels / 2) - 1);

	int jobCount = maxPixels - minPixels;

	// Filter out job batches with no work in them
	if(jobCount == 0) {
		return;
	}

	// + 1 because we go from minPixels to <= maxPixels
	jobCount++;

	jobCount = min(minPixels + jobCount, (spinImageWidthPixels / 2)) - minPixels;

	for(int jobID = 0; jobID < jobCount; jobID++) 
	{

		int jobVertexIndexIndex;
		float jobMinVectorZ;
		float jobMidVectorZ;
		float jobDeltaMinMidZ;
		float jobDeltaMidMaxZ;
		float jobShortDeltaVectorZ;
		float jobShortVectorStartZ;
		float2 jobMinXY;
		float2 jobMidXY;
		float2 jobDeltaMinMidXY;
		float2 jobDeltaMidMaxXY;
		float2 jobShortVectorStartXY;
		float2 jobShortTransformedDelta;

		int jobMinYPixels = minPixels;
		int jobPixelY = jobMinYPixels + jobID;

		jobMinXY = minXY;
		jobMidXY = midXY;

		jobMinVectorZ = minVector.z;
		jobMidVectorZ = midVector.z;

		jobDeltaMinMidZ = deltaMinMid.z;
		jobDeltaMidMaxZ = deltaMidMax.z;

		jobDeltaMinMidXY = deltaMinMidXY;

		jobDeltaMidMaxXY = deltaMidMaxXY;

		jobVertexIndexIndex = settings.vertexIndexIndex;

		// Verified: this should be <=, because it fails for the cube test case
		if (float(jobPixelY) <= jobMidVectorZ)
		{
			// shortVectorStartXY, Bottom: minXY
			jobShortVectorStartXY = jobMinXY;
			// shortVectorStart, Bottom: minVector
			jobShortVectorStartZ = jobMinVectorZ;
			// shortDeltaVector, Bottom: deltaMinMid
			jobShortDeltaVectorZ = jobDeltaMinMidZ;
			// shortTransformedDelta, Bottom: deltaMinMidXY
			jobShortTransformedDelta = jobDeltaMinMidXY;
		}
		else
		{
			// shortVectorStartXY, Top: midXY
			jobShortVectorStartXY = jobMidXY;
			// shortVectorStart, Top: midVector
			jobShortVectorStartZ = jobMidVectorZ;
			// shortDeltaVector, Top: deltaMidMax
			jobShortDeltaVectorZ = jobDeltaMidMaxZ;
			// shortTransformedDelta, Top: deltaMidMaxXY
			jobShortTransformedDelta = jobDeltaMidMaxXY;
		}

		float jobZLevel = float(jobPixelY);
		float jobLongDistanceInTriangle = jobZLevel - jobMinVectorZ;
		float jobLongInterpolationFactor = jobLongDistanceInTriangle / deltaMinMax.z;
		float jobShortDistanceInTriangle = jobZLevel - jobShortVectorStartZ;
		float jobShortInterpolationFactor = (jobShortDeltaVectorZ == 0) ? 1.0f : jobShortDistanceInTriangle / jobShortDeltaVectorZ;
		// Set value to 1 because we want to avoid a zero division, and we define the job Z level to be at its maximum height

		int jobPixelYCoordinate = jobPixelY + (spinImageWidthPixels / 2);
		// Avoid overlap situations, only rasterise is the interpolation factors are valid
		if (jobLongDistanceInTriangle > 0 && jobShortDistanceInTriangle > 0)
		{
			// y-coordinates of both interpolated values are always equal. As such we only need to interpolate that direction once.
			// They must be equal because we have aligned the direction of the horizontal-triangle plane with the x-axis.
			float jobIntersectionY = jobMinXY.y + (jobLongInterpolationFactor * deltaMinMaxXY.y);
			// The other two x-coordinates are interpolated separately.
			float jobIntersection1X = jobShortVectorStartXY.x + (jobShortInterpolationFactor * jobShortTransformedDelta.x);
			float jobIntersection2X = jobMinXY.x + (jobLongInterpolationFactor * deltaMinMaxXY.x);

			float jobIntersection1Distance = length(make_float2(jobIntersection1X, jobIntersectionY));
			float jobIntersection2Distance = length(make_float2(jobIntersection2X, jobIntersectionY));

			// Check < 0 because we omit the case where there is exactly one point with a double intersection
			bool jobHasDoubleIntersection = (jobIntersection1X * jobIntersection2X) < 0;

			// If both values are positive or both values are negative, there is no double intersection.
			// iF the signs of the two values is different, the result will be negative or 0.
			// Having different signs implies the existence of double intersections.
			float jobDoubleIntersectionDistance = abs(jobIntersectionY);

			float jobMinDistance = jobIntersection1Distance < jobIntersection2Distance ? jobIntersection1Distance : jobIntersection2Distance;
			float jobMaxDistance = jobIntersection1Distance > jobIntersection2Distance ? jobIntersection1Distance : jobIntersection2Distance;

			unsigned int jobRowStartPixels = unsigned(floor(jobMinDistance)); // * settings.oneOverPixelSize
			unsigned int jobRowEndPixels = unsigned(floor(jobMaxDistance)); // * settings.oneOverPixelSize

			// Ensure we are only rendering within bounds
			jobRowStartPixels = min((unsigned int)spinImageWidthPixels, max(0, jobRowStartPixels));
			jobRowEndPixels = min((unsigned int)spinImageWidthPixels, jobRowEndPixels);

			size_t jobSpinImageBaseIndex = size_t(jobVertexIndexIndex) * spinImageWidthPixels * spinImageWidthPixels + jobPixelYCoordinate * spinImageWidthPixels;

			// Step 9: Fill pixels
			if (jobHasDoubleIntersection)
			{
				// since this is an absolute value, it can only be 0 or higher.
				int jobDoubleIntersectionStartPixels = int(floor(jobDoubleIntersectionDistance));// * settings.oneOverPixelSize

				// rowStartPixels must already be in bounds, and doubleIntersectionStartPixels can not be smaller than 0.
				// Hence the values in this loop are in-bounds.
#if QSI_PIXEL_DATATYPE == DATATYPE_UNSIGNED_INT || QSI_PIXEL_DATATYPE == DATATYPE_FLOAT32
				for (int jobX = jobDoubleIntersectionStartPixels; jobX < jobRowStartPixels; jobX++)
				{
					// Increment pixel by 2 because 2 intersections occurred.
#if !ENABLE_SHARED_MEMORY_IMAGE
					size_t jobPixelIndex = jobSpinImageBaseIndex + jobX;
					atomicAdd(&(descriptors.content[jobPixelIndex]), 2);
#else
					int jobPixelIndex = jobPixelYCoordinate * spinImageWidthPixels + jobX;
					atomicAdd(&(sharedDescriptorArray[jobPixelIndex]), 2);
#endif

				}
#elif QSI_PIXEL_DATATYPE == DATATYPE_UNSIGNED_SHORT
	#if !ENABLE_SHARED_MEMORY_IMAGE
				int jobBaseIndex = jobSpinImageBaseIndex + jobDoubleIntersectionStartPixels;
				newSpinImagePixelType* descriptorArrayPointer = descriptors.content;
	#else
				int jobBaseIndex = jobPixelYCoordinate * spinImageWidthPixels + jobDoubleIntersectionStartPixels;
				newSpinImagePixelType* descriptorArrayPointer = sharedDescriptorArray;
	#endif
				rasteriseRow(jobBaseIndex, descriptorArrayPointer, jobDoubleIntersectionStartPixels, jobRowStartPixels, SHORT_DOUBLE_ONE_MASK, SHORT_DOUBLE_BOTH_MASK, SHORT_DOUBLE_FIRST_MASK);
#endif
				// Now that we have already covered single intersections in the range minPixels -> doubleIntersectionEndPixels, we move the starting point for the next loop.
				// Not needed because the double intersection range is always smaller than the closest edge point
				//rowStartPixels = doubleIntersectionStartPixels + 1;
			}

#if QSI_PIXEL_DATATYPE == DATATYPE_UNSIGNED_INT || QSI_PIXEL_DATATYPE == DATATYPE_FLOAT32
			// It's imperative the condition of this loop is a < comparison
			for (int jobX = jobRowStartPixels; jobX < jobRowEndPixels; jobX++)
			{
	#if !ENABLE_SHARED_MEMORY_IMAGE
				size_t jobPixelIndex = jobSpinImageBaseIndex + jobX;
				atomicAdd(&(descriptors.content[jobPixelIndex]), 1);
	#else
				int jobPixelIndex = jobPixelYCoordinate * spinImageWidthPixels + jobX;
				atomicAdd(&(sharedDescriptorArray[jobPixelIndex]), 1);
	#endif
			}
#elif QSI_PIXEL_DATATYPE == DATATYPE_UNSIGNED_SHORT
	#if !ENABLE_SHARED_MEMORY_IMAGE
			int jobBaseIndex = jobSpinImageBaseIndex + jobRowStartPixels;
			newSpinImagePixelType* descriptorArrayPointer = descriptors.content;
	#else
			int jobBaseIndex = jobPixelYCoordinate * spinImageWidthPixels + jobRowStartPixels;
			newSpinImagePixelType* descriptorArrayPointer = sharedDescriptorArray;
	#endif
			rasteriseRow(jobBaseIndex, descriptorArrayPointer, jobRowStartPixels, jobRowEndPixels, SHORT_SINGLE_ONE_MASK, SHORT_SINGLE_BOTH_MASK, SHORT_SINGLE_FIRST_MASK);
#endif
		}
	}
}

__launch_bounds__(RASTERISATION_WARP_SIZE) __global__ void generateQuasiSpinImage(
		array<newSpinImagePixelType> descriptors,
		GPURasterisationSettings settings)
{
	// One block x-coordinate per image
	settings.vertexIndexIndex = blockIdx.x;

	// Copying over precalculated values
	settings.spinImageVertex.x = settings.mesh.vertices_x[settings.vertexIndexIndex];
	settings.spinImageVertex.y = settings.mesh.vertices_y[settings.vertexIndexIndex];
	settings.spinImageVertex.z = settings.mesh.vertices_z[settings.vertexIndexIndex];

	settings.spinImageNormal.x = settings.mesh.normals_x[settings.vertexIndexIndex];
	settings.spinImageNormal.y = settings.mesh.normals_y[settings.vertexIndexIndex];
	settings.spinImageNormal.z = settings.mesh.normals_z[settings.vertexIndexIndex];

	assert(__activemask() == 0xFFFFFFFF);

#if ENABLE_SHARED_MEMORY_IMAGE
	assert(__activemask() == 0xFFFFFFFF);

	// Creating a copy of the image in shared memory, then copying it into main memory
	__shared__ newSpinImagePixelType descriptorArrayPointer[spinImageWidthPixels * spinImageWidthPixels];

	// Initialising the values in memory to 0
	for(int i = threadIdx.x; i < spinImageWidthPixels * spinImageWidthPixels; i += RASTERISATION_WARP_SIZE)
	{
		descriptorArrayPointer[i] = 0;
	}

	__syncthreads();
#endif

	const size_t triangleCount = settings.mesh.indexCount / 3;
	for (int triangleIndex = threadIdx.x;
		 triangleIndex < triangleCount;
		 triangleIndex += RASTERISATION_WARP_SIZE)
	{
		float3 vertices[3];

		size_t triangleBaseIndex = 3 * triangleIndex;

		size_t threadTriangleIndex0 = triangleBaseIndex + 0;
		size_t threadTriangleIndex1 = triangleBaseIndex + 1;
		size_t threadTriangleIndex2 = triangleBaseIndex + 2;

		vertices[0].x = settings.mesh.vertices_x[threadTriangleIndex0];
		vertices[0].y = settings.mesh.vertices_y[threadTriangleIndex0];
		vertices[0].z = settings.mesh.vertices_z[threadTriangleIndex0];

		vertices[1].x = settings.mesh.vertices_x[threadTriangleIndex1];
		vertices[1].y = settings.mesh.vertices_y[threadTriangleIndex1];
		vertices[1].z = settings.mesh.vertices_z[threadTriangleIndex1];

		vertices[2].x = settings.mesh.vertices_x[threadTriangleIndex2];
		vertices[2].y = settings.mesh.vertices_y[threadTriangleIndex2];
		vertices[2].z = settings.mesh.vertices_z[threadTriangleIndex2];

	#if ENABLE_SHARED_MEMORY_IMAGE
		rasteriseTriangle(descriptorArrayPointer, vertices, settings);
	#else
		rasteriseTriangle(descriptors, vertices, settings);
	#endif

	}
#if ENABLE_SHARED_MEMORY_IMAGE

#if QSI_PIXEL_DATATYPE == DATATYPE_UNSIGNED_INT

	__syncthreads();
	// Image finished. Copying into main memory
	// Assumption: entire warp processes same spin image
	int jobSpinImageBaseIndex = settings.vertexIndexIndex * spinImageWidthPixels * spinImageWidthPixels;

	for (int i = threadIdx.x; i < spinImageWidthPixels * spinImageWidthPixels; i += RASTERISATION_WARP_SIZE)
	{
		atomicAdd(&descriptors.content[jobSpinImageBaseIndex + i], descriptorArrayPointer[i]);
	}
#elif QSI_PIXEL_DATATYPE == DATATYPE_UNSIGNED_SHORT
	size_t jobSpinImageBaseIndex = size_t(settings.vertexIndexIndex) * spinImageWidthPixels * spinImageWidthPixels;

	unsigned int* integerBasePointer = (unsigned int*)((void*)(descriptors.content + jobSpinImageBaseIndex));
	unsigned int* sharedImageIntPointer = (unsigned int*)((void*)(descriptorArrayPointer));

	// Divide update count by 2 because we update two pixels at a time
	for (int i = threadIdx.x; i < (spinImageWidthPixels * spinImageWidthPixels) / 2; i += RASTERISATION_WARP_SIZE)
	{
		atomicAdd(integerBasePointer + i, *(sharedImageIntPointer + i));
	}
#endif
#endif

}

array<newSpinImagePixelType> generateQuasiSpinImages(DeviceMesh device_mesh, hipDeviceProp_t device_information,
													 float spinImageWidth)
{
	size_t descriptorBufferLength = device_mesh.vertexCount * spinImageWidthPixels * spinImageWidthPixels;
	size_t descriptorBufferSize = sizeof(newSpinImagePixelType) * descriptorBufferLength;

	array<newSpinImagePixelType> device_descriptors;
	checkCudaErrors(hipMalloc(&device_descriptors.content, descriptorBufferSize));

	size_t imageCount = device_mesh.vertexCount;
	device_descriptors.length = imageCount;

	CudaLaunchDimensions valueSetSettings = calculateCudaLaunchDimensions(descriptorBufferLength, device_information);
	setValue<newSpinImagePixelType><< <valueSetSettings.blocksPerGrid, valueSetSettings.threadsPerBlock >> > (device_descriptors.content, descriptorBufferLength, 0);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	auto start = std::chrono::steady_clock::now();

	GPURasterisationSettings generalSettings;
	generalSettings.mesh = device_mesh;

	generateQuasiSpinImage <<<imageCount, RASTERISATION_WARP_SIZE>>> (device_descriptors, generalSettings);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	std::chrono::milliseconds duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start);
	std::cout << "Execution time:" << duration.count() << std::endl;

    array<newSpinImagePixelType> host_descriptors;
	host_descriptors.content = new newSpinImagePixelType[imageCount * spinImageWidthPixels * spinImageWidthPixels];
	host_descriptors.length = imageCount;

	checkCudaErrors(hipMemcpy(host_descriptors.content, device_descriptors.content, descriptorBufferSize, hipMemcpyDeviceToHost));

	hipFree(device_descriptors.content);

	return host_descriptors;
}

