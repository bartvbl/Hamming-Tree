#include "hip/hip_runtime.h"
#include <shapeSearch/gpu/types/DeviceMesh.h>
#include <hip/hip_runtime.h>
#include <shapeSearch/libraryBuildSettings.h>
#include <hiprand/hiprand_kernel.h>
#include <tgmath.h>
#include <assert.h>
#include <iostream>
#include <climits>
#include <cfloat>
#include <chrono>
#include "nvidia/hip/hip_runtime_api.h"
#include "quasiSpinImageSearcher.cuh"



__inline__ __device__ float warpAllReduceSum(float val) {
	for (int mask = warpSize/2; mask > 0; mask /= 2)
		val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
	return val;
}

template<typename pixelType>
__device__ float computeImagePairCorrelation(pixelType* descriptors,
											 pixelType* otherDescriptors,
                                             size_t spinImageIndex,
                                             size_t otherImageIndex,
                                             float averageX, float averageY) {
	float threadSquaredSumX = 0;
	float threadSquaredSumY = 0;
	float threadMultiplicativeSum = 0;

	for (int y = 0; y < spinImageWidthPixels; y++)
	{
		for (int x = threadIdx.x; x < spinImageWidthPixels; x += blockDim.x)
		{
            const size_t spinImageElementCount = spinImageWidthPixels * spinImageWidthPixels;

			pixelType pixelValueX = descriptors[spinImageIndex * spinImageElementCount + (y * spinImageWidthPixels + x)];
			pixelType pixelValueY = otherDescriptors[otherImageIndex * spinImageElementCount + (y * spinImageWidthPixels + x)];

			float deltaX = float(pixelValueX) - averageX;
			float deltaY = float(pixelValueY) - averageY;

			threadSquaredSumX += deltaX * deltaX;
			threadSquaredSumY += deltaY * deltaY;
			threadMultiplicativeSum += deltaX * deltaY;
		}
	}

	float squaredSumX = sqrt(warpAllReduceSum(threadSquaredSumX));
    float squaredSumY = sqrt(warpAllReduceSum(threadSquaredSumY));
    float multiplicativeSum = warpAllReduceSum(threadMultiplicativeSum);

    float correlation = -1;

    // Avoid zero divisions
    if(squaredSumX != 0 && squaredSumY != 0)
    {
        correlation = multiplicativeSum / (squaredSumX * squaredSumY);
    }

    return correlation;
}

template<typename pixelType>
__device__ float computeImageAverage(pixelType* descriptors, size_t spinImageIndex)
{
	const unsigned int spinImageElementCount = spinImageWidthPixels * spinImageWidthPixels;

	float threadPartialSum = 0;

	for (int y = 0; y < spinImageWidthPixels; y++)
	{
		for (int x = threadIdx.x; x < spinImageWidthPixels; x += blockDim.x)
		{
			float pixelValue = float(descriptors[spinImageIndex * spinImageElementCount + (y * spinImageWidthPixels + x)]);
			threadPartialSum += pixelValue;
		}
	}

	return warpAllReduceSum(threadPartialSum) / float(spinImageElementCount);
}

template<typename pixelType>
__global__ void calculateImageAverages(pixelType* images, float* averages) {
	// This kernel assumes one warp per image
	assert(blockDim.x == 32);

	size_t imageIndex = blockIdx.x;

	float average = computeImageAverage<pixelType>(images, imageIndex);

	if(threadIdx.x == 0) {
		averages[imageIndex] = average;
	}
}

template<typename pixelType>
__global__ void generateSearchResults(pixelType* needleDescriptors,
									  pixelType* haystackDescriptors,
									  size_t haystackImageCount,
									  ImageSearchResults* searchResults,
									  float* needleImageAverages,
									  float* haystackImageAverages) {

	size_t needleImageIndex = blockIdx.x;

	// Pearson correlation, which is used as distance measure, means closer to 1 is better
	// We thus initialise the score to the absolute minimum, so that any score is higher.
	static_assert(SEARCH_RESULT_COUNT == 128, "Array initialisation needs to change if search result count is changed");
	size_t threadSearchResultImageIndexes[SEARCH_RESULT_COUNT / 32] = {UINT_MAX, UINT_MAX, UINT_MAX, UINT_MAX};
	float threadSearchResultScores[SEARCH_RESULT_COUNT / 32] = {-FLT_MAX, -FLT_MAX, -FLT_MAX, -FLT_MAX}; // FLT_MIN represents smallest POSITIVE float

	float needleImageAverage = needleImageAverages[needleImageIndex];

	for(size_t haystackImageIndex = 0; haystackImageIndex < haystackImageCount; haystackImageIndex++) {
		float haystackImageAverage = haystackImageAverages[haystackImageIndex];

		float correlation = computeImagePairCorrelation(needleDescriptors,
														haystackDescriptors,
														needleImageIndex,
														haystackImageIndex,
														needleImageAverage,
														haystackImageAverage);

		// Since most images will not make it into the top ranking, we do a quick check to avoid a search
		// This saves a few instructions.
		if(correlation > __shfl_sync(0xFFFFFFFF, threadSearchResultScores[(SEARCH_RESULT_COUNT / 32) - 1], 31)) {

		    // Issue: does not insert correctly in an empty list
		    /*unsigned int leftBound = 0;
			unsigned int rightBound = blockDim.x - 1;
			unsigned int pivotIndex = (leftBound + rightBound) / 2;

			while(leftBound <= rightBound) {
				pivotIndex = (leftBound + rightBound) / 2;
				float pivotThreadValue = __shfl_sync(0xFFFFFFFF, threadSearchResultScore, pivotIndex);
				if(pivotThreadValue < correlation) {
					leftBound = pivotIndex + 1;
				} else if(pivotThreadValue > correlation) {
					rightBound = pivotIndex - 1;
				} else {
					break;
				}
			}*/

            unsigned int foundIndex = 0;
            for(; foundIndex < SEARCH_RESULT_COUNT; foundIndex++) {
                float threadValue = __shfl_sync(0xFFFFFFFF, threadSearchResultScores[foundIndex / 32], foundIndex % 32);
                if(threadValue < correlation) {
                    break;
                }
            }

            int foundThreadIndex = foundIndex % 32;
            int startBlock = foundIndex / 32;
            const int endBlock = (SEARCH_RESULT_COUNT / 32) - 1;
            for(int block = endBlock; block > startBlock; block--) {
				int targetThread = int(threadIdx.x) - 1;
				int targetBlock = block;
				if(targetThread == -1) {
					targetThread = 31;
					targetBlock = block - 1;
				}

				threadSearchResultScores[targetBlock] = __shfl_sync(0xFFFFFFFF, threadSearchResultScores[targetBlock], targetThread);
				threadSearchResultImageIndexes[targetBlock] = __shfl_sync(0xFFFFFFFF, threadSearchResultImageIndexes[targetBlock], targetThread);
            }
			if(threadIdx.x >= foundIndex) {
				int targetThread = int(threadIdx.x) - 1;

				// Shift all values one thread to the right
				threadSearchResultScores[startBlock] = __shfl_sync(0xFFFFFFFF, threadSearchResultScores[startBlock], targetThread);
				threadSearchResultImageIndexes[startBlock] = __shfl_sync(0xFFFFFFFF, threadSearchResultImageIndexes[startBlock], targetThread);

				if(threadIdx.x == foundIndex) {
					threadSearchResultScores[startBlock] = correlation;
					threadSearchResultImageIndexes[startBlock] = haystackImageIndex;
				}
			}

		}
	}

	// Storing search results
	for(int block = 0; block < SEARCH_RESULT_COUNT / 32; block++) {
        searchResults[needleImageIndex].resultIndices[block * SEARCH_RESULT_COUNT + threadIdx.x] = threadSearchResultImageIndexes[block];
        searchResults[needleImageIndex].resultScores[block * SEARCH_RESULT_COUNT + threadIdx.x] = threadSearchResultScores[block];
    }

}

template<typename pixelType>
array<ImageSearchResults> doFindDescriptorsInHaystack(
                                 array<pixelType> device_needleDescriptors,
                                 size_t needleImageCount,
                                 array<pixelType> device_haystackDescriptors,
                                 size_t haystackImageCount)
{
    // Step 1: Compute image averages, since they're constant and are needed for each comparison

	float* device_needleImageAverages;
	float* device_haystackImageAverages;
	checkCudaErrors(hipMalloc(&device_needleImageAverages, needleImageCount * sizeof(float)));
	checkCudaErrors(hipMalloc(&device_haystackImageAverages, haystackImageCount * sizeof(float)));

	std::cout << "\t\tComputing image averages.." << std::endl;
	calculateImageAverages<pixelType><<<needleImageCount, 32>>>(device_needleDescriptors.content, device_needleImageAverages);
	calculateImageAverages<pixelType><<<haystackImageCount, 32>>>(device_haystackDescriptors.content, device_haystackImageAverages);
	checkCudaErrors(hipDeviceSynchronize());

	// Step 2: Perform search

	size_t searchResultBufferSize = needleImageCount * sizeof(ImageSearchResults);
	ImageSearchResults* device_searchResults;
	checkCudaErrors(hipMalloc(&device_searchResults, searchResultBufferSize));

	std::cout << "\t\tPerforming search.." << std::endl;
    auto start = std::chrono::steady_clock::now();

    generateSearchResults<<<needleImageCount, 32>>>(device_needleDescriptors.content,
													device_haystackDescriptors.content,
													haystackImageCount,
													device_searchResults,
													device_needleImageAverages,
													device_haystackImageAverages);
    checkCudaErrors(hipDeviceSynchronize());

    std::chrono::milliseconds duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start);
    std::cout << "\t\t\tExecution time: " << duration.count() << std::endl;

    // Step 3: Copying results to CPU

	array<ImageSearchResults> searchResults;
	searchResults.content = new ImageSearchResults[needleImageCount];
	searchResults.length = needleImageCount;

	checkCudaErrors(hipMemcpy(searchResults.content, device_searchResults, searchResultBufferSize, hipMemcpyDeviceToHost));

	// Cleanup

	hipFree(device_needleImageAverages);
	hipFree(device_haystackImageAverages);
	hipFree(device_searchResults);

	return searchResults;
}

array<ImageSearchResults> findDescriptorsInHaystack(
		array<classicSpinImagePixelType > device_needleDescriptors,
		size_t needleImageCount,
		array<classicSpinImagePixelType > device_haystackDescriptors,
		size_t haystackImageCount) {
	return doFindDescriptorsInHaystack<classicSpinImagePixelType>(device_needleDescriptors, needleImageCount, device_haystackDescriptors, haystackImageCount);
}

array<ImageSearchResults> findDescriptorsInHaystack(
		array<newSpinImagePixelType> device_needleDescriptors,
		size_t needleImageCount,
		array<newSpinImagePixelType > device_haystackDescriptors,
		size_t haystackImageCount) {
	return doFindDescriptorsInHaystack<newSpinImagePixelType>(device_needleDescriptors, needleImageCount, device_haystackDescriptors, haystackImageCount);
}