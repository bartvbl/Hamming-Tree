#include "hip/hip_runtime.h"
#include <shapeSearch/gpu/types/DeviceMesh.h>
#include <hip/hip_runtime_api.h>
#include <shapeSearch/libraryBuildSettings.h>
#include <hiprand/hiprand_kernel.h>
#include <tgmath.h>
#include <assert.h>
#include <iostream>
#include <climits>
#include <cfloat>
#include <chrono>
#include "nvidia/hip/hip_runtime_api.h"
#include "quasiSpinImageSearcher.cuh"

__inline__ __device__ float warpAllReduceSum(float val) {
	for (int mask = warpSize/2; mask > 0; mask /= 2)
		val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
	return val;
}

__device__ float computeImagePairCorrelation(newSpinImagePixelType* descriptors,
                                             newSpinImagePixelType* otherDescriptors,
                                             size_t spinImageIndex,
                                             size_t otherImageIndex,
                                             float averageX, float averageY) {
	float threadSquaredSumX = 0;
	float threadSquaredSumY = 0;
	float threadMultiplicativeSum = 0;

	for (int y = 0; y < spinImageWidthPixels; y++)
	{
		for (int x = threadIdx.x; x < spinImageWidthPixels; x += blockDim.x)
		{
            const size_t spinImageElementCount = spinImageWidthPixels * spinImageWidthPixels;

            newSpinImagePixelType pixelValueX = descriptors[spinImageIndex * spinImageElementCount + (y * spinImageWidthPixels + x)];
            newSpinImagePixelType pixelValueY = otherDescriptors[otherImageIndex * spinImageElementCount + (y * spinImageWidthPixels + x)];

			float deltaX = pixelValueX - averageX;
			float deltaY = pixelValueY - averageY;

			threadSquaredSumX += deltaX * deltaX;
			threadSquaredSumY += deltaY * deltaY;
			threadMultiplicativeSum += deltaX * deltaY;
		}
	}

	float squaredSumX = sqrt(warpAllReduceSum(threadSquaredSumX));
    float squaredSumY = sqrt(warpAllReduceSum(threadSquaredSumY));
    float multiplicativeSum = warpAllReduceSum(threadMultiplicativeSum);

    float correlation = -1;

    // Avoid zero divisions
    if(squaredSumX != 0 && squaredSumY != 0)
    {
        correlation = multiplicativeSum / (squaredSumX * squaredSumY);
    }

    return correlation;
}

__device__ float computeImageAverage(newSpinImagePixelType* descriptors, size_t spinImageIndex)
{
	const unsigned int spinImageElementCount = spinImageWidthPixels * spinImageWidthPixels;

	float threadPartialSum = 0;

	for (int y = 0; y < spinImageWidthPixels; y++)
	{
		for (int x = threadIdx.x; x < spinImageWidthPixels; x += blockDim.x)
		{
			float pixelValue = descriptors[spinImageIndex * spinImageElementCount + (y * spinImageWidthPixels + x)];
			threadPartialSum += pixelValue;
		}
	}

	return warpAllReduceSum(threadPartialSum) / float(spinImageElementCount);
}

__global__ void calculateImageAverages(newSpinImagePixelType* images, float* averages) {
	// This kernel assumes one warp per image
	assert(blockDim.x == 32);

	size_t imageIndex = blockIdx.x;

	float average = computeImageAverage(images, imageIndex);

	if(threadIdx.x == 0) {
		averages[imageIndex] = average;
	}
}

__global__ void generateSearchResults(newSpinImagePixelType* needleDescriptors,
									  newSpinImagePixelType* haystackDescriptors,
									  size_t haystackImageCount,
									  ImageSearchResults* searchResults,
									  float* needleImageAverages,
									  float* haystackImageAverages) {
	// This kernel assumes one warp per triangle
	assert(blockDim.x == 32);

	size_t needleImageIndex = blockIdx.x;

	// Pearson correlation, which is used as distance measure, means closer to 1 is better
	// We thus initialise the score to the absolute minimum, so that any score is higher.
	size_t threadSearchResultImageIndex = UINT_MAX;
	float threadSearchResultScore = -FLT_MAX; // FLT_MIN represents smallest POSITIVE float

	float needleImageAverage = needleImageAverages[needleImageIndex];

	for(size_t haystackImageIndex = 0; haystackImageIndex < haystackImageCount; haystackImageIndex++) {
		float haystackImageAverage = haystackImageAverages[haystackImageIndex];

		float correlation = computeImagePairCorrelation(needleDescriptors,
														haystackDescriptors,
														needleImageIndex,
														haystackImageIndex,
														needleImageAverage,
														haystackImageAverage);

		// Since most images will not make it into the top ranking, we do a quick check to avoid a search
		// This saves a few instructions.
		if(correlation > __shfl_sync(0xFFFFFFFF, threadSearchResultScore, 31)) {

		    // Issue: does not insert correctly in an empty list
		    /*unsigned int leftBound = 0;
			unsigned int rightBound = blockDim.x - 1;
			unsigned int pivotIndex = (leftBound + rightBound) / 2;

			while(leftBound <= rightBound) {
				pivotIndex = (leftBound + rightBound) / 2;
				float pivotThreadValue = __shfl_sync(0xFFFFFFFF, threadSearchResultScore, pivotIndex);
				if(pivotThreadValue < correlation) {
					leftBound = pivotIndex + 1;
				} else if(pivotThreadValue > correlation) {
					rightBound = pivotIndex - 1;
				} else {
					break;
				}
			}*/
		    assert(__activemask() == 0xFFFFFFFF);

            unsigned int foundIndex = 0;
            for(; foundIndex < blockDim.x; foundIndex++) {
                float threadValue = __shfl_sync(0xFFFFFFFF, threadSearchResultScore, foundIndex);
                if(threadValue < correlation) {
                    break;
                }
            }

			// Binary search complete. pivotIndex is now the index at which the found value should be inserted.
			if(threadIdx.x >= foundIndex) {
				// Shift all values one thread to the right
				threadSearchResultScore = __shfl_sync(0xFFFFFFFF, threadSearchResultScore, threadIdx.x - 1);
				threadSearchResultImageIndex = __shfl_sync(0xFFFFFFFF, threadSearchResultImageIndex, threadIdx.x - 1);

				if(threadIdx.x == foundIndex) {
					threadSearchResultScore = correlation;
					threadSearchResultImageIndex = haystackImageIndex;
				}
			}
		}
	}

	// Storing search results
	searchResults[needleImageIndex].resultIndices[threadIdx.x] = threadSearchResultImageIndex;
	searchResults[needleImageIndex].resultScores[threadIdx.x] = threadSearchResultScore;

}

array<ImageSearchResults> findDescriptorsInHaystack(
                                 array<newSpinImagePixelType> device_needleDescriptors,
                                 size_t needleImageCount,
                                 array<newSpinImagePixelType> device_haystackDescriptors,
                                 size_t haystackImageCount)
{
    // Step 1: Compute image averages, since they're constant and are needed for each comparison

	float* device_needleImageAverages;
	float* device_haystackImageAverages;
	checkCudaErrors(hipMalloc(&device_needleImageAverages, needleImageCount * sizeof(float)));
	checkCudaErrors(hipMalloc(&device_haystackImageAverages, haystackImageCount * sizeof(float)));

	std::cout << "\t\tComputing image averages.." << std::endl;
	calculateImageAverages<<<needleImageCount, 32>>>(device_needleDescriptors.content, device_needleImageAverages);
	calculateImageAverages<<<haystackImageCount, 32>>>(device_haystackDescriptors.content, device_haystackImageAverages);
	checkCudaErrors(hipDeviceSynchronize());

	float* debug_needleAverages = new float[needleImageCount];
	float* debug_sampleAverages = new float[haystackImageCount];
	hipMemcpy(debug_needleAverages, device_needleImageAverages, needleImageCount * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(debug_sampleAverages, device_haystackImageAverages, haystackImageCount * sizeof(float), hipMemcpyDeviceToHost);
	for(int i = 0; i < needleImageCount; i++) {
	    std::cout << debug_needleAverages[i] << ", ";
	}
	std::cout << std::endl;
	for(int i = 0; i < haystackImageCount; i++) {
		std::cout << debug_sampleAverages[i] << ", ";
	}
	std::cout << std::endl;
	delete[] debug_needleAverages;
	delete[] debug_sampleAverages;

	// Step 2: Perform search

	size_t searchResultBufferSize = needleImageCount * sizeof(ImageSearchResults);
	ImageSearchResults* device_searchResults;
	checkCudaErrors(hipMalloc(&device_searchResults, searchResultBufferSize));

	std::cout << "\t\tPerforming search.." << std::endl;
    auto start = std::chrono::steady_clock::now();

    generateSearchResults<<<needleImageCount, 32>>>(device_needleDescriptors.content,
													device_haystackDescriptors.content,
													haystackImageCount,
													device_searchResults,
													device_needleImageAverages,
													device_haystackImageAverages);
    checkCudaErrors(hipDeviceSynchronize());

    std::chrono::milliseconds duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start);
    std::cout << "\t\t\tExecution time: " << duration.count() << std::endl;

    // Step 3: Copying results to CPU

	array<ImageSearchResults> searchResults;
	searchResults.content = new ImageSearchResults[needleImageCount];
	searchResults.length = needleImageCount;

	checkCudaErrors(hipMemcpy(searchResults.content, device_searchResults, searchResultBufferSize, hipMemcpyDeviceToHost));

	// Cleanup

	hipFree(device_needleImageAverages);
	hipFree(device_haystackImageAverages);
	hipFree(device_searchResults);

	return searchResults;
}