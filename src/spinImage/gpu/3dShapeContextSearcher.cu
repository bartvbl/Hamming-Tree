#include "hip/hip_runtime.h"
#include <algorithm>
#include "3dShapeContextSearcher.cuh"


const unsigned int warpCount = 16;


__inline__ __device__ float warpAllReduceSum(float val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
    return val;
}

__device__ float compute3DSCPairCorrelationGPU(
        shapeContextBinType* descriptors,
        shapeContextBinType* otherDescriptors,
        size_t descriptorIndex,
        size_t otherDescriptorIndex) {

    for

    return correlation;
}

__global__ void calculateImageAverages(spinImagePixelType* images, float* averages) {
    size_t imageIndex = blockIdx.x;

    // Only support up to 32 warps
    __shared__ float warpSums[32];

    if(threadIdx.x < 32) {
        warpSums[threadIdx.x] = 0;
    }

    __syncthreads();

    const unsigned int spinImageElementCount = spinImageWidthPixels * spinImageWidthPixels;

    float threadPartialSum = 0;

    for (int x = threadIdx.x; x < spinImageElementCount; x += blockDim.x)
    {
        float pixelValue = float(images[imageIndex * spinImageElementCount + x]);
        threadPartialSum += pixelValue;
    }

    float warpSum = warpAllReduceSum(threadPartialSum);

    if(threadIdx.x % 32 == 0) {
        warpSums[threadIdx.x / 32] = warpSum;
    }

    __syncthreads();

    if(threadIdx.x < 32) {
        float threadSum = warpSums[threadIdx.x];
        threadSum = warpAllReduceSum(threadSum);
        if(threadIdx.x == 0) {
            averages[imageIndex] = threadSum / float(spinImageElementCount);
        }
    }
}

__global__ void generateSearchResults(spinImagePixelType* needleDescriptors,
                                      size_t needleImageCount,
                                      spinImagePixelType* haystackDescriptors,
                                      size_t haystackImageCount,
                                      SpinImage::gpu::SpinImageSearchResults* searchResults,
                                      float* needleImageAverages,
                                      float* haystackImageAverages) {

    size_t needleImageIndex = warpCount * blockIdx.x + (threadIdx.x / 32);

    if(needleImageIndex >= needleImageCount) {
        return;
    }

    // Pearson correlation, which is used as distance measure, means closer to 1 is better
    // We thus initialise the score to the absolute minimum, so that any score is higher.
    static_assert(SEARCH_RESULT_COUNT == 128, "Array initialisation needs to change if search result count is changed");
    size_t threadSearchResultImageIndexes[SEARCH_RESULT_COUNT / 32] = {UINT_MAX, UINT_MAX, UINT_MAX, UINT_MAX};
    float threadSearchResultScores[SEARCH_RESULT_COUNT / 32] = {-FLT_MAX, -FLT_MAX, -FLT_MAX, -FLT_MAX}; // FLT_MIN represents smallest POSITIVE float

    const int blockCount = (SEARCH_RESULT_COUNT / 32);

    float needleImageAverage = needleImageAverages[needleImageIndex];

    for(size_t haystackImageIndex = 0; haystackImageIndex < haystackImageCount; haystackImageIndex++) {
        float haystackImageAverage = haystackImageAverages[haystackImageIndex];

        float correlation = computeSpinImagePairCorrelationGPU(
                needleDescriptors,
                haystackDescriptors,
                needleImageIndex,
                haystackImageIndex,
                needleImageAverage,
                haystackImageAverage);

        // Since most images will not make it into the top ranking, we do a quick check to avoid a search
        // This saves a few instructions.
        if(correlation > __shfl_sync(0xFFFFFFFF, threadSearchResultScores[(SEARCH_RESULT_COUNT / 32) - 1], 31)) {
            unsigned int foundIndex = 0;
            for(int block = 0; block < blockCount; block++) {
                bool threadExceeds = threadSearchResultScores[block] < correlation;
                unsigned int bitString = __ballot_sync(0xFFFFFFFF, threadExceeds);
                unsigned int firstSet = __ffs(bitString) - 1;

                if(firstSet < 32) {
                    foundIndex = (block * 32) + (firstSet);
                    break;
                }
            }

            int startBlock = foundIndex / 32;
            const int endBlock = blockCount - 1;
            const int laneID = threadIdx.x % 32;

            // We first shift all values to the right for "full" 32-value blocks
            // Afterwards, we do one final iteration to shift only the values that are
            // block will never be 0, which ensures the loop body does not go out of range
            for(int block = endBlock; block > startBlock; block--) {
                int sourceThread = laneID - 1;
                int sourceBlock = block;

                if(laneID == 0) {
                    sourceThread = 31;
                }
                if(laneID == 31) {
                    sourceBlock = block - 1;
                }

                threadSearchResultScores[block] = __shfl_sync(0xFFFFFFFF, threadSearchResultScores[sourceBlock], sourceThread);
                threadSearchResultImageIndexes[block] = __shfl_sync(0xFFFFFFFF, threadSearchResultImageIndexes[sourceBlock], sourceThread);
            }

            // This shifts over values in the block where we're inserting the new value.
            // As such it requires some more fine-grained control.
            if(laneID >= foundIndex % 32) {
                int targetThread = laneID - 1;

                threadSearchResultScores[startBlock] = __shfl_sync(0xFFFFFFFF, threadSearchResultScores[startBlock], targetThread);
                threadSearchResultImageIndexes[startBlock] = __shfl_sync(0xFFFFFFFF, threadSearchResultImageIndexes[startBlock], targetThread);

                if(laneID == foundIndex % 32) {
                    threadSearchResultScores[startBlock] = correlation;
                    threadSearchResultImageIndexes[startBlock] = haystackImageIndex;
                }
            }

        }
    }


    const unsigned int laneID = threadIdx.x % 32;
    // Storing search results
    for(int block = 0; block < blockCount; block++) {
        searchResults[needleImageIndex].resultIndices[block * 32 + laneID] = threadSearchResultImageIndexes[block];
        searchResults[needleImageIndex].resultScores[block * 32 + laneID] = threadSearchResultScores[block];
    }

}

SpinImage::array<unsigned int> SpinImage::gpu::compute3DSCSearchResultRanks(
        array<shapeContextBinType> device_needleDescriptors,
        size_t needleDescriptorCount,
        array<shapeContextBinType> device_haystackDescriptors,
        size_t haystackDescriptorCount,
        SpinImage::debug::SCSearchRunInfo* runInfo) {

    // Step 1: Compute image averages, since they're constant and are needed for each comparison

    float* device_needleImageAverages;
    float* device_haystackImageAverages;
    checkCudaErrors(hipMalloc(&device_needleImageAverages, needleImageCount * sizeof(float)));
    checkCudaErrors(hipMalloc(&device_haystackImageAverages, haystackImageCount * sizeof(float)));

    std::cout << "\t\tComputing image averages.." << std::endl;
    calculateImageAverages<<<needleImageCount, 32>>>(device_needleDescriptors.content, device_needleImageAverages);
    checkCudaErrors(hipDeviceSynchronize());
    calculateImageAverages<<<haystackImageCount, 32>>>(device_haystackDescriptors.content, device_haystackImageAverages);
    checkCudaErrors(hipDeviceSynchronize());

    // Step 2: Perform search

    size_t searchResultBufferSize = needleImageCount * sizeof(SpinImageSearchResults);
    SpinImageSearchResults* device_searchResults;
    checkCudaErrors(hipMalloc(&device_searchResults, searchResultBufferSize));

    std::cout << "\t\tPerforming search.." << std::endl;
    auto start = std::chrono::steady_clock::now();

    generateSearchResults<<<(needleImageCount / warpCount) + 1, 32 * warpCount>>>(
            device_needleDescriptors.content,
                    needleImageCount,
                    device_haystackDescriptors.content,
                    haystackImageCount,
                    device_searchResults,
                    device_needleImageAverages,
                    device_haystackImageAverages);
    checkCudaErrors(hipDeviceSynchronize());

    std::chrono::milliseconds duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start);
    std::cout << "\t\t\tExecution time: " << duration.count() << std::endl;

    // Step 3: Copying results to CPU

    array<SpinImageSearchResults> searchResults;
    searchResults.content = new SpinImageSearchResults[needleImageCount];
    searchResults.length = needleImageCount;

    checkCudaErrors(hipMemcpy(searchResults.content, device_searchResults, searchResultBufferSize, hipMemcpyDeviceToHost));

    // Cleanup

    hipFree(device_needleImageAverages);
    hipFree(device_haystackImageAverages);
    hipFree(device_searchResults);

    return searchResults;



    SpinImage::array<unsigned int> searchResultIndices = {
            needleDescriptorCount,
            new unsigned int[needleDescriptorCount]
    };
    std::fill(searchResultIndices.content, searchResultIndices.content + needleDescriptorCount, 0);
    return searchResultIndices;
}