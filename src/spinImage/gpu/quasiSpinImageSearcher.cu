#include "hip/hip_runtime.h"
#include <spinImage/gpu/types/DeviceMesh.h>
#include <spinImage/libraryBuildSettings.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <tgmath.h>
#include <assert.h>
#include <iostream>
#include <climits>
#include <cfloat>
#include <chrono>
#include <typeinfo>
#include "nvidia/hip/hip_runtime_api.h"
#include "quasiSpinImageSearcher.cuh"

__inline__ __device__ int warpAllReduceSum(int val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
    return val;
}

const int indexBasedWarpCount = 16;

__device__ int compareQuasiSpinImagePairGPU(
        const quasiSpinImagePixelType* needleImages,
        const size_t needleImageIndex,
        const quasiSpinImagePixelType* haystackImages,
        const size_t haystackImageIndex) {
    int threadScore = 0;
    const int spinImageElementCount = spinImageWidthPixels * spinImageWidthPixels;
    const int laneIndex = threadIdx.x % 32;
    for(int row = laneIndex; row < spinImageWidthPixels; row++) {

        quasiSpinImagePixelType currentNeedlePixelValue =
                needleImages[needleImageIndex * spinImageElementCount + (row * spinImageWidthPixels + laneIndex)];
        quasiSpinImagePixelType currentHaystackPixelValue =
                haystackImages[haystackImageIndex * spinImageElementCount + (row * spinImageWidthPixels + laneIndex)];

        for(int col = laneIndex; col < spinImageWidthPixels - 32; col += 32) {
            quasiSpinImagePixelType nextNeedlePixelValue =
                    needleImages[needleImageIndex * spinImageElementCount + (row * spinImageWidthPixels + col)];
            quasiSpinImagePixelType nextHaystackPixelValue =
                    haystackImages[haystackImageIndex * spinImageElementCount + (row * spinImageWidthPixels + col)];

            quasiSpinImagePixelType nextRankNeedlePixelValue = __shfl_sync(0xFFFFFFFF,
                    // Input value
                                                                           (laneIndex == 0 ? nextNeedlePixelValue : currentNeedlePixelValue),
                    // Target thread
                                                                           (laneIndex == 31 ? 0 : threadIdx.x + 1));
            quasiSpinImagePixelType nextRankHaystackPixelValue = __shfl_sync(0xFFFFFFFF,
                    // Input value
                                                                             (laneIndex == 0 ? nextHaystackPixelValue : currentHaystackPixelValue),
                    // Target thread
                                                                             (laneIndex == 31 ? 0 : threadIdx.x + 1));

            quasiSpinImagePixelType needleDelta = nextRankNeedlePixelValue - currentNeedlePixelValue;
            quasiSpinImagePixelType haystackDelta = nextRankHaystackPixelValue - currentHaystackPixelValue;

            if(needleDelta != 0) {
                threadScore += (needleDelta - haystackDelta) * (needleDelta - haystackDelta);
            }

            currentNeedlePixelValue = nextNeedlePixelValue;
            currentHaystackPixelValue = nextHaystackPixelValue;
        }

        quasiSpinImagePixelType nextRankNeedlePixelValue = __shfl_sync(0xFFFFFFFF, currentNeedlePixelValue, laneIndex + 1);
        quasiSpinImagePixelType nextRankHaystackPixelValue = __shfl_sync(0xFFFFFFFF, currentHaystackPixelValue, laneIndex + 1);

        quasiSpinImagePixelType needleDelta = nextRankNeedlePixelValue - currentNeedlePixelValue;
        quasiSpinImagePixelType haystackDelta = nextRankHaystackPixelValue - currentHaystackPixelValue;

        threadScore += (needleDelta - haystackDelta) * (needleDelta - haystackDelta);
    }

    int imageScore = warpAllReduceSum(threadScore);

    return imageScore;
}

__global__ void computeQuasiSpinImageSearchResultIndices(
        quasiSpinImagePixelType* needleDescriptors,
        quasiSpinImagePixelType* haystackDescriptors,
        size_t haystackImageCount,
        unsigned int* searchResults) {
    size_t needleImageIndex = blockIdx.x;

    __shared__ quasiSpinImagePixelType referenceImage[spinImageWidthPixels * spinImageWidthPixels];
    for(unsigned int index = threadIdx.x; index < spinImageWidthPixels * spinImageWidthPixels; index += blockDim.x) {
        referenceImage[index] = needleDescriptors[spinImageWidthPixels * spinImageWidthPixels * needleImageIndex + index];
    }

    __syncthreads();

    int referenceScore = compareQuasiSpinImagePairGPU(referenceImage, 0, haystackDescriptors, needleImageIndex);

    if(referenceScore == 0) {
        return;
    }

    unsigned int searchResultRank = 0;

    for(size_t haystackImageIndex = threadIdx.x / 32; haystackImageIndex < haystackImageCount; haystackImageIndex += indexBasedWarpCount) {
        if (needleImageIndex == haystackImageIndex) {
            continue;
        }

        int pairScore = compareQuasiSpinImagePairGPU(referenceImage, 0, haystackDescriptors, haystackImageIndex);

        if(pairScore < referenceScore) {
            searchResultRank++;
        }
    }

    // Since we're running multiple warps, we need to add all indices together to get the correct ranks
    if(threadIdx.x % 32 == 0) {
        atomicAdd(&searchResults[needleImageIndex], searchResultRank);
    }
}


array<unsigned int> SpinImage::gpu::computeSearchResultRanks(
        array<quasiSpinImagePixelType> device_needleDescriptors,
        size_t needleImageCount,
        array<quasiSpinImagePixelType> device_haystackDescriptors,
        size_t haystackImageCount,
        SpinImage::debug::QSISearchRunInfo* runInfo) {

    auto executionStart = std::chrono::steady_clock::now();

    size_t searchResultBufferSize = needleImageCount * sizeof(unsigned int);
    unsigned int* device_searchResults;
    checkCudaErrors(hipMalloc(&device_searchResults, searchResultBufferSize));
    checkCudaErrors(hipMemset(device_searchResults, 0, searchResultBufferSize));

    auto searchStart = std::chrono::steady_clock::now();

    computeQuasiSpinImageSearchResultIndices<<<needleImageCount, 32 * indexBasedWarpCount>>>(
            device_needleDescriptors.content,
                    device_haystackDescriptors.content,
                    haystackImageCount,
                    device_searchResults);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    std::chrono::milliseconds searchDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - searchStart);

    array<unsigned int> resultIndices;
    resultIndices.content = new unsigned int[needleImageCount];
    resultIndices.length = needleImageCount;

    checkCudaErrors(hipMemcpy(resultIndices.content, device_searchResults, searchResultBufferSize, hipMemcpyDeviceToHost));

    // Cleanup

    hipFree(device_searchResults);

    std::chrono::milliseconds executionDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - executionStart);

    if(runInfo != nullptr) {
        runInfo->searchExecutionTimeSeconds = double(searchDuration.count()) / 1000.0;
        runInfo->totalExecutionTimeSeconds = double(executionDuration.count()) / 1000.0;
    }

    return resultIndices;
}